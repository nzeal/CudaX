
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

  // int numberOfSMs;
  // cudaDeviceGetAttribute(&numberOfSMs, cudaDevAttrMultiProcessorCount, deviceId);

  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}

