
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__
void initializeElementsTo(int initialValue, int *a, int N) {
  int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (globalIdx < N)
    a[globalIdx] = initialValue;
}

int main()
{
  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  size_t threads_per_block = 256;

  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  for (int i = 0; i < N; ++i)
    {
      if(a[i] != initialValue)
	{
	  printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
	  hipFree(a);
	  exit(1);
	}
    }
  printf("SUCCESS!\n");

  hipFree(a);
}
  
