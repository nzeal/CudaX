// ======================================
//  Vector addition on GPU using CUDA
//  h_ stands things on the host
//  d_a stands things on the device
// ======================================


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void  initVector(double *u, int n, double c) {
  for (int idx=0; idx<n; idx++)
      u[idx] = c;
}

__global__
void vectorAddGPU(double *a, double *b, double *c, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;   // handles the data at this index
  if (idx<n)
    c[idx] = a[idx] + b[idx];
}

void checkResult(double *res, int n)
{
	for(int i = 0; i < n; i++){
		if(res[i] !=3){
			printf("ERROR!\n");
			break;
      }
   }
   printf("\n %f, %f,  %f \n",  res[1] , res[2], res[100]);
   printf("\n Completed Successfully!\n");
}

int main(int argc, char *argv[]) {
// Number of elements 
    double n = 1024; 

// Host Pointers 
    double *h_x, *h_y, *h_z;
    
    size_t bytes = n * sizeof(double);
  
  //Allocate memory on host
    hipHostMalloc( &h_x, bytes, hipHostMallocDefault);
    hipHostMalloc( &h_y, bytes, hipHostMallocDefault);
    hipHostMalloc( &h_z, bytes, hipHostMallocDefault);
    
    initVector((double *) h_x, n, 1.0);
    initVector((double *) h_y, n, 2.0);

  //Allocate memory on device 
   double *d_x, *d_y, *d_z; 

   hipMalloc(&d_x,bytes);
   hipMalloc(&d_y,bytes);
   hipMalloc(&d_z,bytes);
   
   hipMemcpy(d_x, h_x, bytes, hipMemcpyHostToDevice);
   hipMemcpy(d_y, h_y, bytes, hipMemcpyHostToDevice);

   //init block and grid size
   int num_threads = 1<<10;
   int block_size = ( n + num_threads - 1) / num_threads;
   int grid_size  = (int)ceil((float) n /block_size);
   printf("Grid size is %d\n", grid_size);


  // z = u + v
  vectorAddGPU<<<grid_size, block_size>>>(d_x, d_y, d_z, n);
  hipMemcpy(h_z, d_z, bytes, hipMemcpyDeviceToHost);

  // display the results
  checkResult(h_z,n);

  // free memory 
  hipHostFree(h_x); hipHostFree(h_y); hipHostFree(h_z);
  hipFree(d_x); hipFree(d_y); hipFree(d_z);

  return 0;
}
