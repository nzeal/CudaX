// ======================================
//  Vector addition on GPU using CUDA
//  h_ stands things on the host
//  d_a stands things on the device
// ======================================


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void  initVector(double *u, int n, double c) {
  for (int idx=0; idx<n; idx++)
      u[idx] = c;
}

__global__
void vectorAddGPU(double *a, double *b, double *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void checkResult(double *res, int n)
{
	for(int idx = 0; idx < n; idx++){
		if(res[idx] !=3){
			printf("ERROR!\n");
			break;
      }
   }
   printf("\n %f, %f,  %f \n",  res[1] , res[2], res[1023]);
   printf("\n Completed Successfully!\n");
}

int main(int argc, char *argv[]) {
// Number of elements 
    double n = 1024; 

// Host Pointers 
    double *h_x, *h_y, *h_z;

// Device Pointers 
    double *d_x, *d_y,  *d_z;
    
    size_t bytes = n * sizeof(double);
  
  //Allocate memory on host
    h_x = (double*)malloc(bytes);
    h_y = (double*)malloc(bytes);
    h_z = (double*)malloc(bytes);
    
    initVector((double *) h_x, n, 1.0);
    initVector((double *) h_y, n, 2.0);
//    initVector((double *) h_z, n, 0.0);

  //Allocate memory on device 
   hipMalloc(&d_x,bytes);
   hipMalloc(&d_y,bytes);
   hipMalloc(&d_z,bytes);
   
   hipMemcpy(d_x, h_x, bytes, hipMemcpyHostToDevice);
   hipMemcpy(d_y, h_y, bytes, hipMemcpyHostToDevice);

  // z = u + v
  vectorAddGPU<<<n, 1>>>(d_x, d_y, d_z);
  hipMemcpy(h_z, d_z, bytes, hipMemcpyDeviceToHost);

  // display the results
  checkResult(h_z,n);

  // free memory 
  free(h_x); free(h_y); free(h_z);
  hipFree(d_x); hipFree(d_y); hipFree(d_z);

  return 0;
}
