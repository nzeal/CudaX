
#include <hip/hip_runtime.h>
#include <stdio.h>

//  Host function to initialize vector elements. This function
// simply initializes each element to equal its index in the
// vector.
void initWith(float num, float *a, int N) {
  for(int i = 0; i < N; ++i)
    a[i] = num;
}

// Device kernel stores into `result` the sum of each
// same-indexed value of `a` and `b`.
__global__
void addVectorsInto(float *result, float *a, float *b, int N) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
    result[i] = a[i] + b[i];
}

// Host function to confirm values in `vector`. This function
// assumes all values are the same `target` value.
void checkElementsAre(float target, float *vector, int N) {
  for(int i = 0; i < N; i++) {
    if(vector[i] != target) {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main() {
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  int deviceId;
  hipGetDevice(&deviceId);

  int numberOfSMs;
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

